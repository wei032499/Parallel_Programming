#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int *output, int resX, int resY, int maxIterations) {

    int thisX = threadIdx.x + blockIdx.x * blockDim.x;
    int thisY = threadIdx.y + blockIdx.y * blockDim.y;

    if(thisX >= resX || thisY >= resY)
      return;

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int index = thisX + thisY * resX;
    output[index] = mandel(x, y, maxIterations);

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *output = (int *)malloc(resX * resY * sizeof(int));
    int *outputD;

    hipMalloc(&outputD, resX * resY * sizeof(int));

    int blockDimX = 16;
    int blockDimY = 16;

    dim3 dimBlock(blockDimX, blockDimY); // # threads per block
    dim3 dimGrid((int)ceil((double)resX/blockDimX), (int)ceil((double)resY/blockDimY)); // # blocks per grid
    // printf("%d,%d\n",(int)ceil((double)resX/blockDimX), (int)ceil((double)resY/blockDimY));
    
    mandelKernel<<<dimGrid, dimBlock>>>(stepX, stepY, lowerX, lowerY, outputD, resX, resY, maxIterations);

    hipMemcpy(output, outputD, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img, output, resX * resY * sizeof(int));

    free(output);
    hipFree(outputD);
}

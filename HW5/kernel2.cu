#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int pitch, int *output, int resX, int resY, int maxIterations) {

    int thisX = threadIdx.x + blockIdx.x * blockDim.x;
    int thisY = threadIdx.y + blockIdx.y * blockDim.y;

    if(thisX >= resX || thisY >= resY)
        return;

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int index = thisX + thisY * (pitch/sizeof(int));
    output[index] = mandel(x, y, maxIterations);

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *output;
    int *outputD;
    size_t pitch;

   
    // allocate page-locked and mapped memory
    hipError_t err = hipHostAlloc(&output, resX * resY * sizeof(int), hipHostMallocMapped);
    // printf("%s\n",cudaGetErrorString(err));
    
    hipMallocPitch(&outputD, &pitch, resX * sizeof(int), resY);
    

    int blockDimX = 16;
    int blockDimY = 16;

    dim3 dimBlock(blockDimX, blockDimY); // # threads per block
    dim3 dimGrid((int)ceil((double)resX/blockDimX), (int)ceil((double)resY/blockDimY)); // # blocks per grid

    mandelKernel<<<dimGrid, dimBlock>>>(stepX, stepY, lowerX, lowerY, pitch, outputD, resX, resY, maxIterations);

    hipMemcpy2D(output, resX * sizeof(int), outputD, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, output, resX * resY * sizeof(int));
    
    hipHostFree(output);
    hipFree(outputD);
}
